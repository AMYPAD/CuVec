#include "Python.h"
#include "cuhelpers.h"
#include <cstdio>  // printf
#include <sstream> // std::stringstream

void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

bool PyHandleError(hipError_t err, const char *file, int line) {
  std::stringstream ss;
  ss << file << ':' << line << ": " << hipGetErrorString(err);
  std::string s = ss.str();
  if (err != hipSuccess) {
    PyErr_SetString(PyExc_ValueError, s.c_str());
    return false;
  }
  return true;
}
