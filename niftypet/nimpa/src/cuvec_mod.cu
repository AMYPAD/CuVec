/**
 * CUDA C++11 extension vector for Python
 * - Casper da Costa-Luis (https://github.com/casperdcl) 2021
 */
// #include "cuhelpers.h"
#include "cuvec.cuh"
#include <Python.h>
#include <sstream>  // std::stringstream
#include <typeinfo> // typeid

/** functions */
/// required before accessing on host
static PyObject *dev_sync(PyObject *self, PyObject *args) {
  hipDeviceSynchronize();

  Py_INCREF(Py_None);
  return Py_None;
}
static PyMethodDef cuvec_methods[] = {
    {"dev_sync", dev_sync, METH_NOARGS, "Required before accessing cuvec on host."},
    {NULL, NULL, 0, NULL} // Sentinel
};

/** classes */
/// class PyCuVec<T> and PyCuVec_f = PyCuVec<float>
template <class T> struct PyCuVec { PyObject_HEAD CuVec<T> vec; };
/// __init__
template <class T> static int PyCuVec_init(PyCuVec<T> *self, PyObject *args, PyObject *kwds) {
  int length = 0;
  static char *kwlist[2] = {(char *)"length", NULL};
  if (!PyArg_ParseTupleAndKeywords(args, kwds, "|i", kwlist, &length))
    return -1;
  if (length < 0)
    length = 0;
  self->vec.resize(length);
  return 0;
}
/// __del__
template <class T> static void PyCuVec_dealloc(PyCuVec<T> *self) {
  self->vec.clear();
  Py_TYPE(self)->tp_free((PyObject *)self);
}
/// __str__
template <class T> static PyObject *PyCuVec_str(PyCuVec<T> *self) {
  std::stringstream s;
  s << "cuvec.Vector<" << typeid(T).name() << ">[" << self->vec.size() << "]";
  std::string c = s.str();
  PyObject *ret = PyUnicode_FromString(c.c_str());
  return ret;
}
/// buffer interface
static int PyCuVec_getbuffer_f(PyObject *obj, Py_buffer *view, int flags) {
  if (view == NULL) {
    PyErr_SetString(PyExc_ValueError, "NULL view in getbuffer");
    return -1;
  }

  PyCuVec<float> *self = (PyCuVec<float> *)obj;
  Py_ssize_t *shape = (Py_ssize_t *)malloc(sizeof(Py_ssize_t));
  shape[0] = self->vec.size();
  view->buf = (void *)self->vec.data();
  view->obj = (PyObject *)self;
  view->len = self->vec.size() * sizeof(float);
  view->readonly = 0;
  view->itemsize = sizeof(float);
  view->format = (char *)"f"; // float
  view->ndim = 1;
  view->shape = shape;
  view->strides = &view->itemsize;
  view->suboffsets = NULL;
  view->internal = NULL;

  Py_INCREF(self);
  return 0;
}
template <class T> static void PyCuVec_release(PyObject *obj, Py_buffer *view) {
  if (view == NULL) {
    PyErr_SetString(PyExc_ValueError, "NULL view in release");
    return;
  }
  free(view->shape);

  PyCuVec<T> *self = (PyCuVec<T> *)obj;
  Py_DECREF(self);
}
static PyBufferProcs PyCuVec_as_buffer_f = {
    (getbufferproc)PyCuVec_getbuffer_f,
    (releasebufferproc)PyCuVec_release<float>,
};
/// class
static PyTypeObject PyCuVec_f = {
    PyVarObject_HEAD_INIT(NULL, 0) "cuvec.Vector_f", /* tp_name */
    sizeof(PyCuVec<float>),                          /* tp_basicsize */
    0,                                               /* tp_itemsize */
    (destructor)PyCuVec_dealloc<float>,              /* tp_dealloc */
    0,                                               /* tp_print */
    0,                                               /* tp_getattr */
    0,                                               /* tp_setattr */
    0,                                               /* tp_reserved */
    0,                                               /* tp_repr */
    0,                                               /* tp_as_number */
    0,                                               /* tp_as_sequence */
    0,                                               /* tp_as_mapping */
    0,                                               /* tp_hash  */
    0,                                               /* tp_call */
    (reprfunc)PyCuVec_str<float>,                    /* tp_str */
    0,                                               /* tp_getattro */
    0,                                               /* tp_setattro */
    &PyCuVec_as_buffer_f,                            /* tp_as_buffer */
    Py_TPFLAGS_DEFAULT,                              /* tp_flags */
    "cuvec.Vector<f> object",                        /* tp_doc */
    0,                                               /* tp_traverse */
    0,                                               /* tp_clear */
    0,                                               /* tp_richcompare */
    0,                                               /* tp_weaklistoffset */
    0,                                               /* tp_iter */
    0,                                               /* tp_iternext */
    0,                                               /* tp_methods */
    0,                                               /* tp_members */
    0,                                               /* tp_getset */
    0,                                               /* tp_base */
    0,                                               /* tp_dict */
    0,                                               /* tp_descr_get */
    0,                                               /* tp_descr_set */
    0,                                               /* tp_dictoffset */
    (initproc)PyCuVec_init<float>,                   /* tp_init */
};

/// class PyCuVec_d = PyCuVec<double>
/// buffer interface
static int PyCuVec_getbuffer_d(PyObject *obj, Py_buffer *view, int flags) {
  if (view == NULL) {
    PyErr_SetString(PyExc_ValueError, "NULL view in getbuffer");
    return -1;
  }

  PyCuVec<double> *self = (PyCuVec<double> *)obj;
  Py_ssize_t *shape = (Py_ssize_t *)malloc(sizeof(Py_ssize_t));
  shape[0] = self->vec.size();
  view->buf = (void *)self->vec.data();
  view->obj = (PyObject *)self;
  view->len = self->vec.size() * sizeof(double);
  view->readonly = 0;
  view->itemsize = sizeof(double);
  view->format = (char *)"d"; // double
  view->ndim = 1;
  view->shape = shape;
  view->strides = &view->itemsize;
  view->suboffsets = NULL;
  view->internal = NULL;

  Py_INCREF(self);
  return 0;
}
static PyBufferProcs PyCuVec_as_buffer_d = {
    (getbufferproc)PyCuVec_getbuffer_d,
    (releasebufferproc)PyCuVec_release<double>,
};
/// class
static PyTypeObject PyCuVec_d = {
    PyVarObject_HEAD_INIT(NULL, 0) "cuvec.Vector_d", /* tp_name */
    sizeof(PyCuVec<double>),                         /* tp_basicsize */
    0,                                               /* tp_itemsize */
    (destructor)PyCuVec_dealloc<double>,             /* tp_dealloc */
    0,                                               /* tp_print */
    0,                                               /* tp_getattr */
    0,                                               /* tp_setattr */
    0,                                               /* tp_reserved */
    0,                                               /* tp_repr */
    0,                                               /* tp_as_number */
    0,                                               /* tp_as_sequence */
    0,                                               /* tp_as_mapping */
    0,                                               /* tp_hash  */
    0,                                               /* tp_call */
    (reprfunc)PyCuVec_str<double>,                   /* tp_str */
    0,                                               /* tp_getattro */
    0,                                               /* tp_setattro */
    &PyCuVec_as_buffer_d,                            /* tp_as_buffer */
    Py_TPFLAGS_DEFAULT,                              /* tp_flags */
    "cuvec.Vector<d> object",                        /* tp_doc */
    0,                                               /* tp_traverse */
    0,                                               /* tp_clear */
    0,                                               /* tp_richcompare */
    0,                                               /* tp_weaklistoffset */
    0,                                               /* tp_iter */
    0,                                               /* tp_iternext */
    0,                                               /* tp_methods */
    0,                                               /* tp_members */
    0,                                               /* tp_getset */
    0,                                               /* tp_base */
    0,                                               /* tp_dict */
    0,                                               /* tp_descr_get */
    0,                                               /* tp_descr_set */
    0,                                               /* tp_dictoffset */
    (initproc)PyCuVec_init<double>,                  /* tp_init */
};

/** module */
static struct PyModuleDef cuvec_module = {
    PyModuleDef_HEAD_INIT,
    "cuvec", // module
    "CUDA managed memory with python array buffer and C++ std::vector interfaces.",
    -1, // module keeps state in global variables
    cuvec_methods};
PyMODINIT_FUNC PyInit_cuvec(void) {
  Py_Initialize();
  // import_array();  // load NumPy functionality

  PyObject *m = PyModule_Create(&cuvec_module);
  if (m == NULL)
    return NULL;

  // class PyCuVec_f
  PyCuVec_f.tp_new = PyType_GenericNew;
  if (PyType_Ready(&PyCuVec_f) < 0)
    return NULL;
  Py_INCREF(&PyCuVec_f);
  PyModule_AddObject(m, "Vector_f", (PyObject *)&PyCuVec_f);
  // class PyCuVec_d
  PyCuVec_d.tp_new = PyType_GenericNew;
  if (PyType_Ready(&PyCuVec_d) < 0)
    return NULL;
  Py_INCREF(&PyCuVec_d);
  PyModule_AddObject(m, "Vector_d", (PyObject *)&PyCuVec_d);

  return m;
}
