/**
 * Unifying Python/C++/CUDA memory.
 *
 * Python buffered array -> C++11 `std::vector` -> CUDA managed memory.
 *
 * Copyright (2021) Casper da Costa-Luis
 */
#include "Python.h"
#include "pycuvec.cuh" // PyCuVec, PyCuVec_tp
/** functions */
/// required before accessing on host
static PyObject *dev_sync(PyObject *self, PyObject *args) {
  hipDeviceSynchronize();

  Py_INCREF(Py_None);
  return Py_None;
}
static PyMethodDef cuvec_methods[] = {
    {"dev_sync", dev_sync, METH_NOARGS, "Required before accessing cuvec on host."},
    {NULL, NULL, 0, NULL} // Sentinel
};

/** classes */
static PyCuVec_tp<char> Vector_c;
static PyCuVec_tp<signed char> Vector_b;
static PyCuVec_tp<unsigned char> Vector_B;
// #ifdef _Bool
// #endif
static PyCuVec_tp<short> Vector_h;
static PyCuVec_tp<unsigned short> Vector_H;
static PyCuVec_tp<int> Vector_i;
static PyCuVec_tp<unsigned int> Vector_I;
static PyCuVec_tp<long long> Vector_q;
static PyCuVec_tp<unsigned long long> Vector_Q;
static PyCuVec_tp<float> Vector_f;
static PyCuVec_tp<double> Vector_d;

/** module */
static struct PyModuleDef cuvec_module = {
    PyModuleDef_HEAD_INIT,
    "cuvec", // module
    "CUDA unified memory with python array buffer and C++ std::vector interfaces.",
    -1, // module keeps state in global variables
    cuvec_methods};
PyMODINIT_FUNC PyInit_cuvec(void) {
  Py_Initialize();
  // import_array();  // load NumPy functionality

  PyObject *m = PyModule_Create(&cuvec_module);
  if (m == NULL) return NULL;

  if (PyType_Ready(&Vector_c.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_c.tp_obj);
  PyModule_AddObject(m, Vector_c.name.c_str(), (PyObject *)&Vector_c.tp_obj);

  if (PyType_Ready(&Vector_b.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_b.tp_obj);
  PyModule_AddObject(m, Vector_b.name.c_str(), (PyObject *)&Vector_b.tp_obj);

  if (PyType_Ready(&Vector_B.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_B.tp_obj);
  PyModule_AddObject(m, Vector_B.name.c_str(), (PyObject *)&Vector_B.tp_obj);

  // #ifdef _Bool
  // #endif

  if (PyType_Ready(&Vector_h.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_h.tp_obj);
  PyModule_AddObject(m, Vector_h.name.c_str(), (PyObject *)&Vector_h.tp_obj);

  if (PyType_Ready(&Vector_H.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_H.tp_obj);
  PyModule_AddObject(m, Vector_H.name.c_str(), (PyObject *)&Vector_H.tp_obj);

  if (PyType_Ready(&Vector_i.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_i.tp_obj);
  PyModule_AddObject(m, Vector_i.name.c_str(), (PyObject *)&Vector_i.tp_obj);

  if (PyType_Ready(&Vector_I.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_I.tp_obj);
  PyModule_AddObject(m, Vector_I.name.c_str(), (PyObject *)&Vector_I.tp_obj);

  if (PyType_Ready(&Vector_q.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_q.tp_obj);
  PyModule_AddObject(m, Vector_q.name.c_str(), (PyObject *)&Vector_q.tp_obj);

  if (PyType_Ready(&Vector_Q.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_Q.tp_obj);
  PyModule_AddObject(m, Vector_Q.name.c_str(), (PyObject *)&Vector_Q.tp_obj);

  if (PyType_Ready(&Vector_f.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_f.tp_obj);
  PyModule_AddObject(m, Vector_f.name.c_str(), (PyObject *)&Vector_f.tp_obj);

  if (PyType_Ready(&Vector_d.tp_obj) < 0) return NULL;
  Py_INCREF(&Vector_d.tp_obj);
  PyModule_AddObject(m, Vector_d.name.c_str(), (PyObject *)&Vector_d.tp_obj);

  PyObject *author = Py_BuildValue("s", "Casper da Costa-Luis (https://github.com/casperdcl)");
  if (author == NULL) return NULL;
  PyModule_AddObject(m, "__author__", author);

  PyObject *date = Py_BuildValue("s", "2021");
  if (date == NULL) return NULL;
  PyModule_AddObject(m, "__date__", date);

  PyObject *version = Py_BuildValue("s", "0.1.0");
  if (version == NULL) return NULL;
  PyModule_AddObject(m, "__version__", version);

  return m;
}
