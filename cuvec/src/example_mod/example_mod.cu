#include "hip/hip_runtime.h"
/**
 * Example external extension module using CuVec.
 *
 * Copyright (2021) Casper da Costa-Luis
 */
#include "Python.h"
#include "pycuvec.cuh" // PyCuVec
#ifdef CUVEC_DISABLE_CUDA
#include <chrono> // std::chrono
#else
/** functions */
/// dst = src + 1
__global__ void _d_incr(float *dst, float *src, int X, int Y) {
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  if (x >= X) return;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (y >= Y) return;
  dst[y * X + x] = src[y * X + x] + 1;
}
#endif // CUVEC_DISABLE_CUDA
static PyObject *increment2d_f(PyObject *self, PyObject *args, PyObject *kwargs) {
  PyCuVec<float> *dst = NULL;
  PyCuVec<float> *src = NULL;
  static const char *kwds[] = {"src", "output", NULL};
  if (!PyArg_ParseTupleAndKeywords(args, kwargs, "O|O", (char **)kwds, (PyObject **)&src,
                                   (PyObject **)&dst))
    return NULL;
  if (!src) return NULL;
  std::vector<Py_ssize_t> &N = src->shape;
  if (N.size() != 2) {
    PyErr_SetString(PyExc_IndexError, "`src` must be 2D");
    return NULL;
  }

#ifndef CUVEC_DISABLE_CUDA
  hipEvent_t eStart, eAlloc, eKern;
  hipEventCreate(&eStart);
  hipEventCreate(&eAlloc);
  hipEventCreate(&eKern);
  hipEventRecord(eStart);
#else
  auto eStart = std::chrono::steady_clock::now();
#endif

  if (dst) {
    if (N != dst->shape) {
      PyErr_SetString(PyExc_IndexError, "`output` must be same shape as `src`");
      return NULL;
    }
  } else {
    dst = PyCuVec_zeros_like(src);
    if (!dst) return NULL;
  }

#ifndef CUVEC_DISABLE_CUDA
  hipEventRecord(eAlloc);
  dim3 thrds((N[1] + 31) / 32, (N[0] + 31) / 32);
  dim3 blcks(32, 32);
  _d_incr<<<thrds, blcks>>>(dst->vec.data(), src->vec.data(), N[1], N[0]);
  // hipDeviceSynchronize();
  hipEventRecord(eKern);
  hipEventSynchronize(eKern);
  float alloc_ms, kernel_ms;
  hipEventElapsedTime(&alloc_ms, eStart, eAlloc);
  hipEventElapsedTime(&kernel_ms, eAlloc, eKern);
// fprintf(stderr, "%.3f ms, %.3f ms\n", alloc_ms, kernel_ms);
#else
  auto eAlloc = std::chrono::steady_clock::now();
  for (size_t i = 0; i < src->vec.size(); i++) dst->vec[i] = src->vec[i] + 1;
  auto eKern = std::chrono::steady_clock::now();
  double alloc_ms = std::chrono::duration<double, std::milli>(eAlloc - eStart).count();
  double kernel_ms = std::chrono::duration<double, std::milli>(eKern - eAlloc).count();
// fprintf(stderr, "%.3lf ms, %.3lf ms\n", alloc_ms, kernel_ms);
#endif
  return Py_BuildValue("ddO", double(alloc_ms), double(kernel_ms), (PyObject *)dst);
}
static PyMethodDef example_methods[] = {
    {"increment2d_f", (PyCFunction)increment2d_f, METH_VARARGS | METH_KEYWORDS,
     "Args: src, output (optional). Returns: alloc_ms, kernel_ms, src + 1."},
    {NULL, NULL, 0, NULL} // Sentinel
};

/** module */
static struct PyModuleDef example_mod = {PyModuleDef_HEAD_INIT,
                                         "example_mod", // module
                                         "Example external module.",
                                         -1, // module keeps state in global variables
                                         example_methods};
PyMODINIT_FUNC PyInit_example_mod(void) {
  Py_Initialize();
  return PyModule_Create(&example_mod);
}
