#include "hip/hip_runtime.h"
/**
 * Example external SWIG extension module using CuVec.
 *
 * Copyright (2021) Casper da Costa-Luis
 */
#include "cuvec.cuh" // SwigCuVec, SwigCuVec_new
#include <stdexcept> // std::length_error
/// dst = src + 1
__global__ void _d_incr(float *dst, float *src, int X, int Y) {
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  if (x >= X) return;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  if (y >= Y) return;
  dst[y * X + x] = src[y * X + x] + 1;
}
SwigCuVec<float> *increment2d_f(SwigCuVec<float> &src, SwigCuVec<float> *output, bool timing) {
  auto &N = src.shape;
  if (N.size() != 2) throw std::length_error("`src` must be 2D");

  hipEvent_t eStart, eAlloc, eKern;
  hipEventCreate(&eStart);
  hipEventCreate(&eAlloc);
  hipEventCreate(&eKern);
  hipEventRecord(eStart);
  if (!output)
    output = SwigCuVec_new<float>(N);
  else if (N != output->shape)
    throw std::length_error("`output` must be same shape as `src`");
  hipEventRecord(eAlloc);
  dim3 thrds((N[1] + 31) / 32, (N[0] + 31) / 32);
  dim3 blcks(32, 32);
  _d_incr<<<thrds, blcks>>>(output->vec.data(), src.vec.data(), N[1], N[0]);
  cuvec::HandleError(hipGetLastError(), __FILE__, __LINE__);
  // hipDeviceSynchronize();
  hipEventRecord(eKern);
  hipEventSynchronize(eKern);
  float alloc_ms, kernel_ms;
  hipEventElapsedTime(&alloc_ms, eStart, eAlloc);
  hipEventElapsedTime(&kernel_ms, eAlloc, eKern);
  // fprintf(stderr, "%.3f ms, %.3f ms\n", alloc_ms, kernel_ms);
  if (timing) {
    // hack: store times in first two elements of output
    output->vec[0] = alloc_ms;
    output->vec[1] = kernel_ms;
  }
  return output;
}
